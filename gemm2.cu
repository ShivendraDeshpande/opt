#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <math.h>

#define POLYBENCH_TIME 1

#include "gemm.cuh"
#include "../../common/polybench.h"
#include "../../common/polybenchUtilFuncts.h"

#define GPU_DEVICE 0

//define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

/* Declared constant values for ALPHA and BETA (same as values in PolyBench 2.0) */
#define ALPHA 32412.0f
#define BETA 2123.0f

#define RUN_ON_CPU

#define FP64_RATIO 0.25  // Fraction of work to be done with FP64

void gemm(int ni, int nj, int nk, DATA_TYPE alpha, DATA_TYPE beta,
	 DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nk),
	 DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj),
	 DATA_TYPE POLYBENCH_2D(C,NI,NJ,ni,nj))
{
	int i,j,k;

	for (i = 0; i < _PB_NI; i++)
	{
    		for (j = 0; j < _PB_NJ; j++)
    		{
			C[i][j] *= beta;

			for (k = 0; k < _PB_NK; ++k)
			{
	  			C[i][j] += alpha * A[i][k] * B[k][j];
			}
      		}
	}
}

void init(int ni, int nj, int nk, DATA_TYPE* alpha, DATA_TYPE* beta,
	DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nk),
	DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj),
	DATA_TYPE POLYBENCH_2D(C,NI,NJ,ni,nj))
{
	int i, j;

	*alpha = 32412;
	*beta = 2123;

  	for (i = 0; i < ni; i++)
	{
    		for (j = 0; j < nk; j++)
		{
      			A[i][j] = ((DATA_TYPE) i*j) / NI;
		}
	}

  	for (i = 0; i < nk; i++)
	{
    		for (j = 0; j < nj; j++)
		{
      			B[i][j] = ((DATA_TYPE) i*j) / NI;
		}
	}

  	for (i = 0; i < ni; i++)
	{
    		for (j = 0; j < nj; j++)
		{
      			C[i][j] = ((DATA_TYPE) i*j) / NI;
		}
	}
}


void compareResults(int ni, int nj, DATA_TYPE POLYBENCH_2D(C,NI,NJ,ni,nj), DATA_TYPE POLYBENCH_2D(C_outputFromGpu,NI,NJ,ni,nj))
{
	int i, j, fail;
	fail = 0;

	// Compare CPU and GPU outputs
	for (i=0; i < ni; i++)
	{
		for (j=0; j < nj; j++)
		{
			if (percentDiff(C[i][j], C_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD)
			{
				fail++;
			}
		}
	}

	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

void GPU_argv_init()
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
	printf("setting device %d with name %s\n",GPU_DEVICE,deviceProp.name);
	hipSetDevice( GPU_DEVICE );
}


__global__ void gemm_kernel(int ni, int nj, int nk, float alpha, float beta, float *a, float *b, float *c) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i < ni) && (j < nj)) {
        float sum_fp32 = 0.0f;
        double sum_fp64 = 0.0; // Added FP64 accumulator

        // Hybrid FP32/FP64 computation
        for (int k = 0; k < nk; k++) {
            if (k < (int)(nk * FP64_RATIO)) {  // Use FP64 for the first FP64_RATIO fraction of the loop
                sum_fp64 += (double)a[i * nk + k] * (double)b[k * nj + j];  // Cast to double for FP64 calculation
            } else {
                sum_fp32 += a[i * nk + k] * b[k * nj + j]; // Use FP32 for the rest
            }
        }

        // Combine the results (convert FP64 back to FP32)
        c[i * nj + j] = beta * c[i * nj + j] + alpha * (sum_fp32 + (float)sum_fp64);
    }
}


void gemmCuda(int ni, int nj, int nk, DATA_TYPE alpha, DATA_TYPE beta,
	DATA_TYPE POLYBENCH_2D(A,NI,NK,ni,nk),
	DATA_TYPE POLYBENCH_2D(B,NK,NJ,nk,nj),
	DATA_TYPE POLYBENCH_2D(C,NI,NJ,ni,nj),
	DATA_TYPE POLYBENCH_2D(C_outputFromGpu,NI,NJ,ni,nj))
{
	DATA_TYPE *A_gpu;
	DATA_TYPE *B_gpu;
	DATA_TYPE *C_gpu;

	hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NK);
	hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NK * NJ);
	hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NI * NJ);

	hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
	hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((size_t)(ceil( ((float)NI)/ ((float)block.x) )),(size_t)(ceil( ((float)NJ)/ ((float)block.y) )));

	/* Start timer. */
  	polybench_start_instruments;

	gemm_kernel<<< grid, block >>>(ni, nj, nk, alpha, beta, A_gpu, B_gpu, C_gpu);
	hipDeviceSynchronize();

	/* Stop and print timer. */
	printf("GPU Time in seconds:\n");
  	polybench_stop_instruments;
 	polybench_print_instruments;

	hipMemcpy(C_outputFromGpu, C_gpu, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyDeviceToHost);

	hipFree(A_gpu);
	hipFree(B_gpu);
	hipFree(C_gpu);
}


/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static
void print_array(int ni, int nj,
		 DATA_TYPE POLYBENCH_2D(C,NI,NJ,ni,nj))
{
  int i, j;

  for (i = 0; i < ni; i++)
    for (j = 0; j < nj; j++) {
	fprintf (stderr, DATA_PRINTF_MODIFIER, C[i][j]);
	if ((i * ni + j) % 20 == 0) fprintf (stderr, "\n");
    }
  fprintf (stderr, "\n");
}


int main(int argc, char *argv[])
{
	/* Retrieve problem size. */
	int ni = NI;
	int nj = NJ;
	int nk = NK;

	/* Variable declaration/allocation. */
	DATA_TYPE alpha;
	DATA_TYPE beta;
	POLYBENCH_2D_ARRAY_DECL(A,DATA_TYPE,NI,NK,ni,nk);
	POLYBENCH_2D_ARRAY_DECL(B,DATA_TYPE,NK,NJ,nk,nj);
	POLYBENCH_2D_ARRAY_DECL(C,DATA_TYPE,NI,NJ,ni,nj);
	POLYBENCH_2D_ARRAY_DECL(C_outputFromGpu,DATA_TYPE,NI,NJ,ni,nj);

	init(ni, nj, nk, &alpha, &beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C));

	GPU_argv_init();

	gemmCuda(ni, nj, nk, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));


	#ifdef RUN_ON_CPU

		/* Start timer. */
	  	polybench_start_instruments;

		gemm(ni, nj, nk, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C));

		/* Stop and print timer. */
		printf("CPU Time in seconds:\n");
  		polybench_stop_instruments;
	 	polybench_print_instruments;

		compareResults(ni, nj, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));

	#else //print output to stderr so no dead code elimination

		print_array(ni, nj, POLYBENCH_ARRAY(C_outputFromGpu));

	#endif //RUN_ON_CPU


	POLYBENCH_FREE_ARRAY(A);
	POLYBENCH_FREE_ARRAY(B);
	POLYBENCH_FREE_ARRAY(C);
	POLYBENCH_FREE_ARRAY(C_outputFromGpu);

    	return 0;
}

#include "../../common/polybench.c"