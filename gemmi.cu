#include "hip/hip_runtime.h"
/**
 * gemm.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define POLYBENCH_TIME 1

#include "gemm.cuh"
#include "../../common/polybench.h"
#include "../../common/polybenchUtilFuncts.h"

#define GPU_DEVICE 0

// define the error threshold for the results "not matching"
#define PERCENT_DIFF_ERROR_THRESHOLD 0.05

/* Declared constant values for ALPHA and BETA (same as values in PolyBench 2.0) */
#define ALPHA 32412.0f
#define BETA 2123.0f

#define RUN_ON_CPU

void gemm(int ni, int nj, int nk, DATA_TYPE alpha, DATA_TYPE beta, DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk),
          DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj), DATA_TYPE POLYBENCH_2D(C, NI, NJ, ni, nj))
{
    int i, j, k;

    for (i = 0; i < _PB_NI; i++)
    {
        for (j = 0; j < _PB_NJ; j++)
        {
            C[i][j] *= beta;

            for (k = 0; k < _PB_NK; ++k)
            {
                C[i][j] += alpha * A[i][k] * B[k][j];
            }
        }
    }
}

void init(int ni, int nj, int nk, DATA_TYPE *alpha, DATA_TYPE *beta, DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk),
          DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj), DATA_TYPE POLYBENCH_2D(C, NI, NJ, ni, nj))
{
    int i, j;

    *alpha = 32412;
    *beta = 2123;

    for (i = 0; i < ni; i++)
    {
        for (j = 0; j < nk; j++)
        {
            A[i][j] = ((DATA_TYPE)i * j) / NI;
        }
    }

    for (i = 0; i < nk; i++)
    {
        for (j = 0; j < nj; j++)
        {
            B[i][j] = ((DATA_TYPE)i * j) / NI;
        }
    }

    for (i = 0; i < ni; i++)
    {
        for (j = 0; j < nj; j++)
        {
            C[i][j] = ((DATA_TYPE)i * j) / NI;
        }
    }
}

void compareResults(int ni, int nj, DATA_TYPE POLYBENCH_2D(C, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(C_outputFromGpu, NI, NJ, ni, nj))
{
    int i, j, fail;
    fail = 0;

    // Compare CPU and GPU outputs
    for (i = 0; i < ni; i++)
    {
        for (j = 0; j < nj; j++)
        {
            if (percentDiff(C[i][j], C_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD)
            {
                fail++;
            }
        }
    }

    // Print results
    printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

void GPU_argv_init()
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, GPU_DEVICE);
    printf("setting device %d with name %s\n", GPU_DEVICE, deviceProp.name);
    hipSetDevice(GPU_DEVICE);
}

// Modified kernel to utilize both FP32 and FP64 compute units
__global__ void gemm_kernel_optimized(int ni, int nj, int nk, DATA_TYPE alpha, DATA_TYPE beta,
                                      DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *c)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i < _PB_NI) && (j < _PB_NJ))
    {
        // Apply beta scaling with FP32 cores
        c[i * NJ + j] *= beta;

        // Split the iterations between FP32 and FP64 cores
        int k;
        double fp64_sum = 0.0; // Use FP64 for accumulation

        // Process the first half using FP64 operations
        for (k = 0; k < _PB_NK / 2; k++)
        {
            // Explicitly perform FP64 computation
            fp64_sum += (double)alpha * (double)a[i * NK + k] * (double)b[k * NJ + j];
        }

        // Process the second half using FP32 operations
        DATA_TYPE fp32_sum = 0.0f;
        for (k = _PB_NK / 2; k < _PB_NK; k++)
        {
            // Standard FP32 computation
            fp32_sum += alpha * a[i * NK + k] * b[k * NJ + j];
        }

        // Combine the results
        c[i * NJ + j] += (DATA_TYPE)fp64_sum + fp32_sum;
    }
}

// Add this function to your existing code as a replacement for the original gemm_kernel
void gemmCuda(int ni, int nj, int nk, DATA_TYPE alpha, DATA_TYPE beta,
              DATA_TYPE POLYBENCH_2D(A, NI, NK, ni, nk), DATA_TYPE POLYBENCH_2D(B, NK, NJ, nk, nj),
              DATA_TYPE POLYBENCH_2D(C, NI, NJ, ni, nj), DATA_TYPE POLYBENCH_2D(C_outputFromGpu, NI, NJ, ni, nj))
{
    DATA_TYPE *A_gpu;
    DATA_TYPE *B_gpu;
    DATA_TYPE *C_gpu;

    hipMalloc((void **)&A_gpu, sizeof(DATA_TYPE) * NI * NK);
    hipMalloc((void **)&B_gpu, sizeof(DATA_TYPE) * NK * NJ);
    hipMalloc((void **)&C_gpu, sizeof(DATA_TYPE) * NI * NJ);

    hipMemcpy(A_gpu, A, sizeof(DATA_TYPE) * NI * NK, hipMemcpyHostToDevice);
    hipMemcpy(B_gpu, B, sizeof(DATA_TYPE) * NK * NJ, hipMemcpyHostToDevice);
    hipMemcpy(C_gpu, C, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyHostToDevice);

    dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
    dim3 grid((size_t)(ceil(((float)NI) / ((float)block.x))), (size_t)(ceil(((float)NJ) / ((float)block.y))));

    /* Start timer. */
    polybench_start_instruments;

    // Use the optimized kernel instead
    gemm_kernel_optimized<<<grid, block>>>(ni, nj, nk, alpha, beta, A_gpu, B_gpu, C_gpu);
    hipDeviceSynchronize();

    /* Stop and print timer. */
    printf("GPU Time in seconds (Optimized):\n");
    polybench_stop_instruments;
    polybench_print_instruments;

    hipMemcpy(C_outputFromGpu, C_gpu, sizeof(DATA_TYPE) * NI * NJ, hipMemcpyDeviceToHost);

    hipFree(A_gpu);
    hipFree(B_gpu);
    hipFree(C_gpu);
}

/* DCE code. Must scan the entire live-out data.
   Can be used also to check the correctness of the output. */
static void print_array(int ni, int nj,
                        DATA_TYPE POLYBENCH_2D(C, NI, NJ, ni, nj))
{
    int i, j;

    for (i = 0; i < ni; i++)
        for (j = 0; j < nj; j++)
        {
            fprintf(stderr, DATA_PRINTF_MODIFIER, C[i][j]);
            if ((i * ni + j) % 20 == 0)
                fprintf(stderr, "\n");
        }
    fprintf(stderr, "\n");
}

int main(int argc, char *argv[])
{
    /* Retrieve problem size. */
    int ni = NI;
    int nj = NJ;
    int nk = NK;

    /* Variable declaration/allocation. */
    DATA_TYPE alpha;
    DATA_TYPE beta;
    POLYBENCH_2D_ARRAY_DECL(A, DATA_TYPE, NI, NK, ni, nk);
    POLYBENCH_2D_ARRAY_DECL(B, DATA_TYPE, NK, NJ, nk, nj);
    POLYBENCH_2D_ARRAY_DECL(C, DATA_TYPE, NI, NJ, ni, nj);
    POLYBENCH_2D_ARRAY_DECL(C_outputFromGpu, DATA_TYPE, NI, NJ, ni, nj);

    init(ni, nj, nk, &alpha, &beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C));

    GPU_argv_init();

    gemmCuda(ni, nj, nk, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));

#ifdef RUN_ON_CPU

    /* Start timer. */
    polybench_start_instruments;

    gemm(ni, nj, nk, alpha, beta, POLYBENCH_ARRAY(A), POLYBENCH_ARRAY(B), POLYBENCH_ARRAY(C));

    /* Stop and print timer. */
    printf("CPU Time in seconds:\n");
    polybench_stop_instruments;
    polybench_print_instruments;

    compareResults(ni, nj, POLYBENCH_ARRAY(C), POLYBENCH_ARRAY(C_outputFromGpu));

#else // print output to stderr so no dead code elimination

    print_array(ni, nj, POLYBENCH_ARRAY(C_outputFromGpu));

#endif // RUN_ON_CPU

    POLYBENCH_FREE_ARRAY(A);
    POLYBENCH_FREE_ARRAY(B);
    POLYBENCH_FREE_ARRAY(C);
    POLYBENCH_FREE_ARRAY(C_outputFromGpu);

    return 0;
}

#include "../../common/polybench.c"
